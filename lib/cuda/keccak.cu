#include "hip/hip_runtime.h"
//
//  =============== KECCAK part on nVidia GPU ======================
//
// The keccak512 (SHA-3) is used in the PBKDF2 for scrypt-jane coins
// in place of the SHA2 based PBKDF2 used in scrypt coins.
//
// NOTE: compile this .cu module for compute_20,sm_20 with --maxrregcount=64
//

#include <map>

#include "api_internal.h"
#include "cuda_helper.h"

#include "keccak.h"
#include "salsa_kernel.h"

// define some error checking macros
#define DELIMITER '/'
#define __FILENAME__ ( strrchr(__FILE__, DELIMITER) != NULL ? strrchr(__FILE__, DELIMITER)+1 : __FILE__ )

#undef checkCudaErrors
#define checkCudaErrors(gpuId, x) \
{ \
	hipGetLastError(); \
	x; \
	hipError_t err = hipGetLastError(); \
	if (err != hipSuccess && !abort_flag) \
		applog(LOG_ERR, "GPU #%d: hipError_t %d (%s) (%s line %d)\n", gpuId, err, hipGetErrorString(err), __FILENAME__, __LINE__); \
}

// from salsa_kernel.cu
extern std::map<int, uint32_t *> context_idata[2];
extern std::map<int, uint32_t *> context_odata[2];
extern std::map<int, hipStream_t> context_streams[2];
extern std::map<int, uint32_t *> context_hash[2];

#ifndef ROTL64
#define ROTL64(a,b) (((a) << (b)) | ((a) >> (64 - b)))
#endif

// CB
#define U32TO64_LE(p) \
	(((uint64_t)(*p)) | (((uint64_t)(*(p + 1))) << 32))

#define U64TO32_LE(p, v) \
	*p = (uint32_t)((v)); *(p+1) = (uint32_t)((v) >> 32);

static __device__ void mycpy64(uint32_t *d, const uint32_t *s)
{
#pragma unroll 16
	for (int k = 0; k < 16; ++k) {
		d[k] = s[k];
	}
}

// ---------------------------- BEGIN keccak functions ------------------------------------

#define KECCAK_HASH "Keccak-512"
#define SCRYPT_HASH_DIGEST_SIZE 64
#define SCRYPT_KECCAK_F 1600
#define SCRYPT_KECCAK_C (SCRYPT_HASH_DIGEST_SIZE * 8 * 2) /* 1024 */
#define SCRYPT_KECCAK_R (SCRYPT_KECCAK_F - SCRYPT_KECCAK_C) /* 576 */
#define SCRYPT_HASH_BLOCK_SIZE (SCRYPT_KECCAK_R / 8)

typedef struct keccak_hash_state_t {
	uint64_t state[25];					// 25*2
	uint32_t buffer[72 / 4];			// 72
} keccak_hash_state;

__device__ void statecopy0(keccak_hash_state *d, keccak_hash_state *s)
{
#pragma unroll 25
	for (int i = 0; i < 25; ++i) {
		d->state[i] = s->state[i];
	}
}

static const uint64_t host_keccak_round_constants[24] = {
	0x0000000000000001ull, 0x0000000000008082ull,
	0x800000000000808aull, 0x8000000080008000ull,
	0x000000000000808bull, 0x0000000080000001ull,
	0x8000000080008081ull, 0x8000000000008009ull,
	0x000000000000008aull, 0x0000000000000088ull,
	0x0000000080008009ull, 0x000000008000000aull,
	0x000000008000808bull, 0x800000000000008bull,
	0x8000000000008089ull, 0x8000000000008003ull,
	0x8000000000008002ull, 0x8000000000000080ull,
	0x000000000000800aull, 0x800000008000000aull,
	0x8000000080008081ull, 0x8000000000008080ull,
	0x0000000080000001ull, 0x8000000080008008ull
};

__constant__ uint64_t c_keccak_round_constants[24];
__constant__ uint32_t c_data[20];

#define U8TO32_LE(p)                                            \
	(((uint32_t)((p)[0])      ) | ((uint32_t)((p)[1]) <<  8) |  \
	 ((uint32_t)((p)[2]) << 16) | ((uint32_t)((p)[3]) << 24))

#define U32TO8_LE(p, v)                                           \
	(p)[0] = (uint8_t)((v)      ); (p)[1] = (uint8_t)((v) >>  8); \
	(p)[2] = (uint8_t)((v) >> 16); (p)[3] = (uint8_t)((v) >> 24);

#define U8TO64_LE(p)                                                  \
	(((uint64_t)U8TO32_LE(p)) | ((uint64_t)U8TO32_LE((p) + 4) << 32))

#define U64TO8_LE(p, v)                        \
	U32TO8_LE((p),     (uint32_t)((v)      )); \
	U32TO8_LE((p) + 4, (uint32_t)((v) >> 32));

__device__
void keccak_block(keccak_hash_state *S, const uint32_t *in)
{
	uint64_t *s = S->state, t[5], u[5], v, w;

	/* absorb input */
#pragma unroll 9
	for (int i = 0; i < 72 / 8; i++, in += 2) {
		s[i] ^= U32TO64_LE(in);
	}

	for (int i = 0; i < 24; i++) {
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[0] = t[4] ^ ROTL64(t[1], 1);
		u[1] = t[0] ^ ROTL64(t[2], 1);
		u[2] = t[1] ^ ROTL64(t[3], 1);
		u[3] = t[2] ^ ROTL64(t[4], 1);
		u[4] = t[3] ^ ROTL64(t[0], 1);

		/* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
		s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
		s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
		s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
		s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
		s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

		/* rho pi: b[..] = rotl(a[..], ..) */
		v = s[1];
		s[1] = ROTL64(s[6], 44);
		s[6] = ROTL64(s[9], 20);
		s[9] = ROTL64(s[22], 61);
		s[22] = ROTL64(s[14], 39);
		s[14] = ROTL64(s[20], 18);
		s[20] = ROTL64(s[2], 62);
		s[2] = ROTL64(s[12], 43);
		s[12] = ROTL64(s[13], 25);
		s[13] = ROTL64(s[19], 8);
		s[19] = ROTL64(s[23], 56);
		s[23] = ROTL64(s[15], 41);
		s[15] = ROTL64(s[4], 27);
		s[4] = ROTL64(s[24], 14);
		s[24] = ROTL64(s[21], 2);
		s[21] = ROTL64(s[8], 55);
		s[8] = ROTL64(s[16], 45);
		s[16] = ROTL64(s[5], 36);
		s[5] = ROTL64(s[3], 28);
		s[3] = ROTL64(s[18], 21);
		s[18] = ROTL64(s[17], 15);
		s[17] = ROTL64(s[11], 10);
		s[11] = ROTL64(s[7], 6);
		s[7] = ROTL64(s[10], 3);
		s[10] = ROTL64(v, 1);

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = s[0]; w = s[1]; s[0] ^= (~w) & s[2]; s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & v; s[4] ^= (~v) & w;
		v = s[5]; w = s[6]; s[5] ^= (~w) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; s[8] ^= (~s[9]) & v; s[9] ^= (~v) & w;
		v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
		v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
		v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

		/* iota: a[0,0] ^= round constant */
		s[0] ^= c_keccak_round_constants[i];
	}
}

__device__
void keccak_hash_init(keccak_hash_state *S)
{
#pragma unroll 25
	for (int i = 0; i < 25; ++i) {
		S->state[i] = 0ULL;
	}
}

// assuming there is no leftover data and exactly 72 bytes are incoming
// we can directly call into the block hashing function
__device__ void keccak_hash_update72(keccak_hash_state *S, const uint32_t *in)
{
	keccak_block(S, in);
}

__device__ void keccak_hash_update4(keccak_hash_state *S, const uint32_t *in)
{
	*S->buffer = *in;
}

__device__ void keccak_hash_update64(keccak_hash_state *S, const uint32_t *in)
{
	mycpy64(S->buffer, in);
}

__device__
void keccak_hash_finish4(keccak_hash_state *S, uint32_t *hash)
{
	S->buffer[4 / 4] = 0x01;
#pragma unroll
	for (int i = 4 / 4 + 1; i < 72 / 4; ++i) {
		S->buffer[i] = 0;
	}
	S->buffer[72 / 4 - 1] |= 0x80000000U;
	keccak_block(S, (const uint32_t*)S->buffer);

#pragma unroll 8
	for (int i = 0; i < 64; i += 8) {
		U64TO32_LE((&hash[i / 4]), S->state[i / 8]);
	}
}

__device__
void keccak_hash_finish64(keccak_hash_state *S, uint32_t *hash)
{
	S->buffer[64 / 4] = 0x01;
#pragma unroll
	for (int i = 64 / 4 + 1; i < 72 / 4; ++i) {
		S->buffer[i] = 0;
	}
	S->buffer[72 / 4 - 1] |= 0x80000000U;
	keccak_block(S, (const uint32_t*)S->buffer);

#pragma unroll 8
	for (int i = 0; i < 64; i += 8) {
		U64TO32_LE((&hash[i / 4]), S->state[i / 8]);
	}
}

__device__
uint8_t keccak_hash_1_finish64(keccak_hash_state *S)
{
	S->buffer[64 / 4] = 0x01;
#pragma unroll
	for (int i = 64 / 4 + 1; i < 72 / 4; ++i) {
		S->buffer[i] = 0;
	}
	S->buffer[72 / 4 - 1] |= 0x80000000U;
	keccak_block(S, (const uint32_t*)S->buffer);

	return S->state[0];
}

__device__
uint8_t keccak_hash_finish(keccak_hash_state *S, uint32_t buffered, uint32_t *hash)
{
	uint32_t i = buffered / 4;
	S->buffer[i] = 0x01;

	for (i = i + 1; i < 72 / 4; ++i) {
		S->buffer[i] = 0;
	}

	S->buffer[72 / 4 - 1] |= 0x80000000U;
	keccak_block(S, (const uint32_t*)S->buffer);

#pragma unroll 8
	for (int i = 0; i < 64; i += 8) {
		U64TO32_LE((&hash[i / 4]), S->state[i / 8]);
	}
}

// ---------------------------- END keccak functions ------------------------------------

// ---------------------------- BEGIN PBKDF2 functions ------------------------------------

typedef struct pbkdf2_hmac_state_t {
	keccak_hash_state inner, outer;
} pbkdf2_hmac_state;

/* hmac */

__device__
void pbkdf2_hmac_init72(pbkdf2_hmac_state *st, const uint32_t *key)
{
	uint32_t pad[72 / 4] = { 0 };

	keccak_hash_init(&st->inner);
	keccak_hash_init(&st->outer);

#pragma unroll 18
	for (int i = 0; i < 72 / 4; i++) {
		pad[i] = key[i];
	}

	/* inner = (key ^ 0x36) */
	/* h(inner || ...) */
#pragma unroll 18
	for (int i = 0; i < 72 / 4; i++) {
		pad[i] ^= 0x36363636U;
	}
	keccak_hash_update72(&st->inner, pad);

	/* outer = (key ^ 0x5c) */
	/* h(outer || ...) */
#pragma unroll 18
	for (int i = 0; i < 72 / 4; i++) {
		pad[i] ^= 0x6a6a6a6aU;
	}
	keccak_hash_update72(&st->outer, pad);
}

__device__ void pbkdf2_hmac_update4(pbkdf2_hmac_state *st, const uint32_t *m)
{
	/* h(inner || m...) */
	keccak_hash_update4(&st->inner, m);
}

__device__
uint32_t pbkdf2_hmac_update(pbkdf2_hmac_state *st, const uint32_t *m, uint32_t length)
{
	/* h(inner || m...) */
	while (length >= 72) {
		keccak_hash_update72(&st->inner, m);
		length -= 72;
		m += 74 / 4;
	}

	if (length > 0)  ((uint64_t*)st->inner.buffer)[0] = ((const uint64_t*)m)[0];
	if (length > 8)  ((uint64_t*)st->inner.buffer)[1] = ((const uint64_t*)m)[1];
	if (length > 16) ((uint64_t*)st->inner.buffer)[2] = ((const uint64_t*)m)[2];
	if (length > 24) ((uint64_t*)st->inner.buffer)[3] = ((const uint64_t*)m)[3];
	if (length > 32) ((uint64_t*)st->inner.buffer)[4] = ((const uint64_t*)m)[4];
	if (length > 40) ((uint64_t*)st->inner.buffer)[5] = ((const uint64_t*)m)[5];
	if (length > 48) ((uint64_t*)st->inner.buffer)[6] = ((const uint64_t*)m)[6];
	if (length > 56) ((uint64_t*)st->inner.buffer)[7] = ((const uint64_t*)m)[7];

	return length;
}

__device__
uint32_t pbkdf2_hmac_buffer_update4(pbkdf2_hmac_state *st, const uint32_t m, uint32_t buffered)
{
	st->inner.buffer[buffered / 4] = m;
	return buffered + 4;
}

__device__ void pbkdf2_hmac_finish4(pbkdf2_hmac_state *st, uint32_t *mac)
{
	/* h(inner || m) */
	uint32_t innerhash[16];
	keccak_hash_finish4(&st->inner, innerhash);

	/* h(outer || h(inner || m)) */
	keccak_hash_update64(&st->outer, innerhash);
	keccak_hash_finish64(&st->outer, mac);
}

__device__ uint8_t pbkdf2_hmac_finish(pbkdf2_hmac_state *st, uint32_t buffered)
{
	/* h(inner || m) */
	uint32_t innerhash[16];
	keccak_hash_finish(&st->inner, buffered, innerhash);

	/* h(outer || h(inner || m)) */
	keccak_hash_update64(&st->outer, innerhash);
	return keccak_hash_1_finish64(&st->outer);
}

__device__ void pbkdf2_statecopy0(pbkdf2_hmac_state *d, pbkdf2_hmac_state *s)
{
	statecopy0(&d->inner, &s->inner);
	statecopy0(&d->outer, &s->outer);
}

// ---------------------------- END PBKDF2 functions ------------------------------------

#define U32TO8_BE(p, v)                                           \
	(p)[0] = (uint8_t)((v) >> 24); (p)[1] = (uint8_t)((v) >> 16); \
	(p)[2] = (uint8_t)((v) >>  8); (p)[3] = (uint8_t)((v)      );

__global__ __launch_bounds__(128)
void cuda_pre_keccak512(uint32_t *g_idata, uint64_t nonce, uint32_t r)
{
	uint32_t i, blocks;
	uint32_t data[20];
	uint8_t be[4];

	const uint32_t thread = (blockIdx.x * blockDim.x) + threadIdx.x;
	nonce   += thread;
	g_idata += thread * 32 * r;

#pragma unroll
	for (int i = 0; i < 19; i++) {
		data[i] = c_data[i];
	}
	((uint64_t*)data)[4] = nonce;

	pbkdf2_hmac_state hmac_pw;
	uint32_t bytes = r * 128;

	/* hmac(password, ...) */
	pbkdf2_hmac_init72(&hmac_pw, data);

	pbkdf2_hmac_state work;
	uint32_t ti[16];

	blocks = ((uint32_t)bytes + (SCRYPT_HASH_DIGEST_SIZE - 1)) / SCRYPT_HASH_DIGEST_SIZE;
	for (i = 1; i <= blocks; i++) {
		/* U1 = hmac(password, salt || be(i)) */
		uint32_t be = cuda_swab32(i);
		pbkdf2_statecopy0(&work, &hmac_pw);
		pbkdf2_hmac_update4(&work, &be);
		pbkdf2_hmac_finish4(&work, ti);
		mycpy64(g_idata, ti);

		g_idata += SCRYPT_HASH_DIGEST_SIZE / sizeof(uint32_t);
		bytes -= SCRYPT_HASH_DIGEST_SIZE;
	}
}

__global__ __launch_bounds__(128)
void cuda_post_keccak512(uint32_t *g_odata, uint8_t *labels, uint64_t nonce, uint32_t r)
{
	uint32_t data[20];

	const uint32_t thread = (blockIdx.x * blockDim.x) + threadIdx.x;
	labels  += thread;
	g_odata += thread * 32 * r;
	nonce   += thread;

#pragma unroll
	for (int i = 0; i < 19; i++) {
		data[i] = c_data[i];
	}
	((uint64_t*)data)[4] = nonce;

	pbkdf2_hmac_state hmac_pw;

	/* hmac(password, ...) */
	pbkdf2_hmac_init72(&hmac_pw, data);

	/* hmac(password, salt...) */
	uint32_t buffered = pbkdf2_hmac_update(&hmac_pw, g_odata, 128 * r);

	/* U1 = hmac(password, salt || be(i)) */
	uint32_t be = 0x01000000U;//cuda_swab32(1);
	buffered = pbkdf2_hmac_buffer_update4(&hmac_pw, be, buffered);
	*labels = pbkdf2_hmac_finish(&hmac_pw, buffered);
}

//
// callable host code to initialize constants and to call kernels
//

extern "C" void prepare_keccak512(_cudaState *cudaState, const uint8_t *host_pdata, const uint32_t pdata_size)
{
	if (!cudaState->keccak_inited) {
		checkCudaErrors(cudaState->cuda_id, hipMemcpyToSymbol(HIP_SYMBOL(c_keccak_round_constants), host_keccak_round_constants, sizeof(host_keccak_round_constants), 0, hipMemcpyHostToDevice));
		cudaState->keccak_inited = true;
	}
	checkCudaErrors(cudaState->cuda_id, hipMemcpyToSymbol(HIP_SYMBOL(c_data), host_pdata, pdata_size, 0, hipMemcpyHostToDevice));
}

extern "C" void pre_keccak512(_cudaState *cudaState, int stream, uint64_t nonce, int throughput, uint32_t r)
{
	dim3 block(128);
	dim3 grid((throughput + 127) / 128);

	cuda_pre_keccak512 << <grid, block, 0, cudaState->context_streams[stream] >> >(cudaState->context_idata[stream], nonce, r);
}

extern "C" void post_keccak512(_cudaState *cudaState, int stream, uint64_t nonce, int throughput, uint32_t r)
{
	dim3 block(128);
	dim3 grid((throughput + 127) / 128);

	cuda_post_keccak512 << <grid, block, 0, cudaState->context_streams[stream] >> >((uint32_t *)cudaState->context_odata[stream], cudaState->context_labels[stream], nonce, r);
}
